#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "tipos.h"
#include "kmer.cuh"
#include "mpi.h"

void lersequencia (char *buf, int bufsizeInt, int rank, int *aux)
{

   int i = bufsizeInt;
   int j = 0;

   while ( i > 0)
   {
      if (buf[i] == '>')
      {
         *aux = i;
         j = j + 1;
         break;
      }
      else //serve para qnd o processo não tem inicio de sequencia
      {
         *aux = bufsizeInt;
         i = i - 1;
      }
   }
}

int main (int argc, char *argv[])
{
   int rank, nprocs; // posição do processo e total de processo no comunicador
   char *str_rank, *arq_inic, *arq_saida, hostname[MPI_MAX_PROCESSOR_NAME];
   int k, bufsize_new=0, bufsizeInt=0, i=0, dest=0;
   double bufsizeDouble=0;
   lint chunkSize = 4096;
   char *buf;
   int aux=0, tag1=1;
   /*para o novo deslocamento de cada view
   vet_send e vet_recv: primeira posição ultima visualizão do processo, segunda posição quantidade de dados que o processo terá que acresentar em sua nova visualização*/
   int inic_view_atual=0;
   int vet_send[2]= {0,0}, vet_recv[2]={0,0};

   if (argc < 4)
   {
      printf("Usage: ./kmer [dataset.fasta] [k] [outputfile] <chunkSize: Default 4096>");
      return 1;
   }
   if (argc == 5)
      chunkSize = atoi(argv[4]);

   MPI_Init(&argc, &argv);
   MPI_File fh;
   MPI_Status status;
   MPI_Offset filesize;
   MPI_Datatype filetype;
   MPI_Request r;
   MPI_Comm_rank(MPI_COMM_WORLD, &rank);
   MPI_Comm_size(MPI_COMM_WORLD, &nprocs);

   arq_inic = (char*)malloc(256*sizeof(char));
   arq_saida = (char*)malloc(256*sizeof(char));
   strcpy (arq_inic, argv[1]);
   strcpy (arq_saida, argv[2]);
   k = atoi (argv[3]);
   str_rank = (char*)malloc(sizeof(char));

   MPI_File_open(MPI_COMM_WORLD, arq_inic, MPI_MODE_RDONLY, MPI_INFO_NULL, &fh);

   MPI_File_get_size(fh, &filesize);

   bufsizeDouble = ((double)filesize / nprocs);
   bufsizeInt = ceil(bufsizeDouble);
   buf = (char *)malloc(bufsizeInt * sizeof(char));

   //printf("bufsize: %d \n", bufsizeInt);
   //printf("filesize: %d \n", filesize);

   MPI_Barrier(MPI_COMM_WORLD);

   MPI_File_set_view(fh, rank*bufsizeInt, MPI_CHAR, filetype, "native", MPI_INFO_NULL);

   MPI_File_read_at_all(fh, rank*bufsizeInt, buf, bufsizeInt, MPI_CHAR, &status);

   lersequencia(buf, bufsizeInt, rank, &aux);

   //comunicação para saber quantos dados a mais irei ler (vcopy) e quantidade que o processo anterior temninava sua visualização (end_view_prsant)
   vet_send[0]= (rank*bufsizeInt)+bufsizeInt-1;
   vet_send[1]= bufsizeInt - aux;
   //printf("vet_send [0]: %d, vet_send [1]: %d rank:%d\n\n", vet_send[0], vet_send[1], rank);

   MPI_Irecv(vet_recv, 2, MPI_INT, MPI_ANY_SOURCE, tag1, MPI_COMM_WORLD, &r);
   if(rank != nprocs-1)
   {
      dest = rank + 1;
      MPI_Isend(vet_send, 2, MPI_INT, dest, tag1, MPI_COMM_WORLD, &r);
   }

   MPI_Barrier(MPI_COMM_WORLD);

   //printf("vet_recv [0]: %d, vet_recv [1]: %d rank:%d\n\n", vet_recv[0], vet_recv[1], rank);

   //Inicio da nova view
   inic_view_atual = vet_recv[0] - vet_recv[1];
   //printf("inicio da minha view agora: %d, rank: %d\n", inic_view_atual, rank);

   //calculo para saber novo tamanho do buffer, a partir dos valores de recevi e enviei
   if(rank != nprocs-1)
   {
      bufsize_new = (bufsizeInt+vet_recv[1])-vet_send[1];
   }
   else
   {
      bufsize_new = (bufsizeInt+vet_recv[1])-0;
   }
   printf("tam buf agr: %d, rank: %d\n", bufsize_new, rank);

   //Início da nova visualização
   free(buf);

   buf = (char *)malloc(bufsize_new * sizeof(char));

   if(rank == 0)
   {
      MPI_File_read_at_all(fh, rank*bufsize_new, buf, bufsize_new-1, MPI_CHAR, &status);
   }
   else
   {
      MPI_File_read_at_all(fh, inic_view_atual+1, buf, bufsize_new, MPI_CHAR, &status);
   }

   /*if(rank == 0)
   {
      for(i=0; i<bufsize_new; i++)
      {
         printf("rank: %d, read: %c i: %d\n", rank, buf[i], i);
      }
   }*/

   MPI_Barrier(MPI_COMM_WORLD);

   sprintf (str_rank, "%d", rank);
   strcat(arq_saida, "_");
   strcat(arq_saida, str_rank);
   strcat(arq_saida, ".out");

   cfrk(buf, arq_saida, k, chunkSize, rank, bufsize_new);
   MPI_File_close(&fh);
   MPI_Finalize();
}
