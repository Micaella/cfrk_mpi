#include "hip/hip_runtime.h"
#include <stdio.h>
#include <pthread.h>
#include <math.h>
#include <stdint.h>
#include "kmer.cuh"
#include "tipos.h"
#include "fastaIO.h"
#include "string.h"

void DeviceInfo(FILE *outputfile, uint8_t device)
{
   hipDeviceProp_t prop;

   hipGetDeviceProperties(&prop, device);

   fprintf(outputfile, "\n\n***** Device information *****\n\n");

   fprintf(outputfile, "\tId: %d\n", device);
   fprintf(outputfile, "\tName: %s\n", prop.name);
   fprintf(outputfile, "\tTotal global memory: %ld\n", prop.totalGlobalMem);
   fprintf(outputfile, "\tMax grid size: %d, %d, %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
   fprintf(outputfile, "\tMax thread dim: %d, %d, %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
   fprintf(outputfile, "\tWarp size: %d\n", prop.warpSize);
   fprintf(outputfile, "\tMax threads per multiprocessor: %d\n", prop.maxThreadsPerMultiProcessor);

   fprintf(outputfile, "\n************************************\n\n");
}

int SelectDevice(int devCount)
{

   int i, device = 0;
   hipDeviceProp_t prop[devCount];

   if (devCount > 0)
   {
      for (i = 0; i < devCount; i++)
      {
         hipGetDeviceProperties(&prop[i], i);
      }

      for (i = 0; i < devCount; i++)
      {
         if (prop[i].totalGlobalMem > prop[device].totalGlobalMem)
         {
            device = i;
         }
      }
   }
   else
      return 0;

return device;
}

struct read* SelectChunk(struct read *rd, ushort chunkSize, ushort it, lint gnS, lint *nS, lint gnN, lint *nN)
{
   struct read *chunk;
   int i;
   lint length = 0;

   // Size to be allocated
   for (i = 0; i < chunkSize; i++)
   {
      int id = chunkSize*it + i;
      if (id > gnS-1)
      {
         break;
      }
      length += rd->length[id]+1;
   }

   hipHostMalloc((void**)&chunk, sizeof(struct read));
   hipHostMalloc((void**)&chunk->data, sizeof(short)*length);
   hipHostMalloc((void**)&chunk->length, sizeof(int)*chunkSize);
   hipHostMalloc((void**)&chunk->start, sizeof(int)*chunkSize);

   // Copy rd->data to chunk->data
   lint start = rd->start[chunkSize*it];
   lint end = start + length;
   for (i = start; i < end; i++)
   {
      chunk->data[i-start] = rd->data[i];
   }

   chunk->length[0] = rd->length[chunkSize*it];
   chunk->start[0] = 0;
   // Copy start and length
   for (i = 1; i < chunkSize; i++)
   {
      int id = chunkSize*it + i;
      chunk->length[i] = rd->length[id];
      chunk->start[i] = chunk->start[i-1]+(chunk->length[i-1]+1);
   }

   *nN = length;
   *nS = chunkSize;
return chunk;
}

int cfrk(char *dataset, char *dataout, int k, lint chunkSize)
{

   int device;
   lint gnN, gnS, nS, nN;
   int devCount;

   FILE *outputfile;
   outputfile = fopen(dataout, "a");
   
   hipDeviceReset();
   
   hipGetDeviceCount(&devCount);
   device = SelectDevice(devCount);
   DeviceInfo(outputfile, device);

   //fprintf(outputfile, "\ndataset: %s, k: %d, chunkSize: %d\n", dataset, k, chunkSize);
   printf("\ndataset: %s, k: %d, chunkSize: %d\n", dataset, k, chunkSize);

   lint st = time(NULL);
   puts("\n\n\t\tReading seqs!!!");
   struct read *rd;
   hipHostMalloc((void**)&rd, sizeof(struct read));
   ReadFASTASequences(dataset, &gnN, &gnS, rd, 1);
   fprintf(outputfile, "\nnS: %ld, nN: %ld\n", gnS, gnN);
   lint et = time(NULL);

   fprintf(outputfile, "\n\t\tReading time: %ld\n", (et-st));

   int nChunk = floor(gnS/chunkSize);
   struct read *chunk;
   printf("passei aqui 1 \n");
   for (int i = 0; i < nChunk; i++)
   {
      chunk = SelectChunk(rd, chunkSize, i, gnS, &nS, gnN, &nN);
      kmer_main(chunk, outputfile, nN, nS, k, device);
      hipFree(chunk->data);
      hipFree(chunk->length);
      hipFree(chunk->start);
      hipFree(chunk);
   }
   printf("passei aqui 2 \n");
   int chunkRemain = abs(gnS - (nChunk*chunkSize));
   chunk = SelectChunk(rd, chunkRemain, nChunk, gnS, &nS, gnN, &nN);
   fprintf(outputfile, "\nnS: %ld, nN: %ld, chunkRemain: %d\n", nS, nN, chunkRemain);
   kmer_main(chunk, outputfile, nN, nS, k, device);

   fclose(outputfile);

return 0;
}
